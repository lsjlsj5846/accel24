#include "hip/hip_runtime.h"
#include "namegen.h"
#include "util.h"

#include <cassert>
#include <math.h>
#include <vector>
#include <string.h>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// You can modify the data structure as you want
struct Tensor {

  Tensor(std::vector<int> shape_) {
    // No initalization for new Tensors
    ndim = shape_.size();
    for (size_t i = 0; i < ndim; i++) {
      shape[i] = shape_[i];
    }

    size_t n = num_elem();
    buf = (float *)malloc(n * sizeof(float));
    set_zero();
    CHECK_CUDA(hipMalloc(&buf_gpu, sizeof(float) * n));
    // buf = (float *)malloc(n * sizeof(float));
  }

  /* Alloc memory */
  Tensor(std::vector<int> shape_, bool _init) {
    ndim = shape_.size();
    for (size_t i = 0; i < ndim; i++) {
      shape[i] = shape_[i];
    }

    size_t n = num_elem();
    buf = (float *)malloc(n * sizeof(float));
    set_zero();
    CHECK_CUDA(hipMalloc(&buf_gpu, sizeof(float) * n));
    if (_init == true){
      CHECK_CUDA(hipMemcpy(buf_gpu, buf, sizeof(float)*n, 
    hipMemcpyHostToDevice));
    }
    // buf = (float *)malloc(n * sizeof(float));
  }

  /* Alloc memory and copy */
  Tensor(std::vector<int> shape_, float *buf_) {
    ndim = shape_.size();
    for (size_t i = 0; i < ndim; i++) {
      shape[i] = shape_[i];
    }

    size_t n = num_elem();
    CHECK_CUDA(hipMalloc(&buf_gpu, sizeof(float) * n));
    CHECK_CUDA(hipMemcpy(buf_gpu, buf_, sizeof(float)*n, 
    hipMemcpyHostToDevice));
    // memcpy(buf, buf_, n * sizeof(float));
  }

  ~Tensor() {
    // if (buf_gpu != nullptr)
    //   CHECK_CUDA(hipFree(buf_gpu));
    if (buf != nullptr)
      free(buf);
  }

  void set_zero() {
    size_t n = num_elem();
    for (size_t i = 0; i < n; i++)
      buf[i] = 0.0;
  }

  size_t num_elem() {
    size_t sz = 1;
    for (size_t i = 0; i < ndim; i++)
      sz *= shape[i];
    return sz;
  }

  // Pointer to data
  float *buf_gpu = nullptr;
  float *buf = nullptr;

  // Shape of tensor, from outermost dimension to innermost dimension.
  // e.g., {{1.0, -0.5, 2.3}, {4.3, 5.6, -7.8}} => shape = {2, 3}
  size_t ndim = 0;
  size_t shape[4];
};

/* Network parameters */
Tensor *character_embedding;
Tensor *W_ir0, *W_iz0, *W_in0, *W_ir1, *W_iz1, *W_in1;
Tensor *W_hr0, *W_hz0, *W_hn0, *W_hr1, *W_hz1, *W_hn1;
Tensor *b_ir0, *b_iz0, *b_in0, *b_ir1, *b_iz1, *b_in1;
Tensor *b_hr0, *b_hz0, *b_hn0, *b_hr1, *b_hz1, *b_hn1;
Tensor *W_fc, *b_fc;
Tensor *rfloats;

/* input, activations, output */
Tensor *input, *emb_out;
Tensor *hidden0, *hidden1;
Tensor *r0, *r1, *z0, *z1, *n0, *n1, *f, *char_prob;
Tensor *rtmp00, *rtmp01, *rtmp02, *rtmp03, *rtmp04;
Tensor *rtmp10, *rtmp11, *rtmp12, *rtmp13, *rtmp14;
Tensor *ztmp00, *ztmp01, *ztmp02, *ztmp03, *ztmp04;
Tensor *ztmp10, *ztmp11, *ztmp12, *ztmp13, *ztmp14;
Tensor *ntmp00, *ntmp01, *ntmp02, *ntmp03, *ntmp04, *ntmp05;
Tensor *ntmp10, *ntmp11, *ntmp12, *ntmp13, *ntmp14, *ntmp15;
Tensor *htmp00, *htmp01, *htmp02;
Tensor *htmp10, *htmp11, *htmp12;
Tensor *ftmp0;

/* Operations */

/*
 * Embedding
 * input: [1] (scalar)
 * weight: [NUM_CHAR x EMBEDDING_DIM]
 * output: [EMBEDDING_DIM]
 */
void embedding(Tensor *input, Tensor *weight, Tensor *output) {
  size_t n = weight->shape[1];
  for (size_t i = 0; i < n; i++) {
    int x = (int)input->buf[0];
    output->buf[i] = weight->buf[x * n + i];
  }
}

/*
 * Elementwise addition
 * input1: [*]
 * input2: [*] (same shape as input1)
 * output: [*] (same shape as input1)
 */
void elemwise_add(Tensor *input1, Tensor *input2, Tensor *output) {
  size_t sn = input1->num_elem();
  for (size_t i = 0; i < sn; i++) {
    output->buf[i] = input1->buf[i] + input2->buf[i];
  }
}

/*
 * Elementwise (1-x)
 * input: [*]
 * output: [*] (same shape as input)
 */
void elemwise_oneminus(Tensor *input, Tensor *output) {
  size_t n = input->num_elem();
  for (size_t i = 0; i < n; i++) {
    float x = input->buf[i];
    output->buf[i] = 1.0 - x;
  }
}

/*
 * Elementwise multiplication
 * input1: [*]
 * input2: [*] (same shape as input1)
 * output: [*] (same shape as input1)
 */
void elemwise_mul(Tensor *input1, Tensor *input2, Tensor *output) {
  size_t sn = input1->num_elem();
  for (size_t i = 0; i < sn; i++) {
    output->buf[i] = input1->buf[i] * input2->buf[i];
  }
}

/*
 * Elementwise tanh(x)
 * input: [*]
 * output: [*] (same shape as input)
 */
void elemwise_tanh(Tensor *input, Tensor *output) {
  size_t n = input->num_elem();
  for (size_t i = 0; i < n; i++) {
    float x = input->buf[i];
    output->buf[i] = tanhf(x);
  }
}

/*
 * Elementwise Sigmoid 1 / (1 + exp(-x))
 * input: [*]
 * output: [*] (same shape as input)
 */
void elemwise_sigmoid(Tensor *input, Tensor *output) {
  size_t n = input->num_elem();
  for (size_t i = 0; i < n; i++) {
    float x = input->buf[i];
    output->buf[i] = 1.0 / (1.0 + expf(-x));
  }
}

// __device__ float _gpu_sigmoid(float x){
//   return 1.0 / (1.0 + expf(-x));
// }

__device__ float _gpu_tanh(float x){
  return tanhf(x);
}
/*
 * SGEMV
 * input1: [N x K]
 * input2: [K]
 * output: [N]
 */
void matvec(Tensor *input1, Tensor *input2, Tensor *output) {
  size_t N_ = input1->shape[0];
  size_t K_ = input1->shape[1];
  for (size_t i = 0; i < N_; i++) {
    float c = 0.0;
    for (size_t j = 0; j < K_; j++) {
      c += input1->buf[i * K_ + j] * input2->buf[j];
    }
    output->buf[i] = c;
  }
}

/*
 * SGEMM
 * input1: [M x K]
 * input2: [K x N]
 * output: [M x N]
 */
void matmul(Tensor *input1, Tensor *input2, Tensor *output) {
  size_t M_ = input1->shape[0];
  size_t K_ = input1->shape[1];
  size_t N_ = input2->shape[1];
  for (size_t i = 0; i < M_; i++) {
    for (size_t j = 0; j < N_; j++) {
      float c = 0.0;
      for (size_t k = 0; k < K_; k++) {
        c += input1->buf[i * K_ + k] * input2->buf[k * N_ + j];
      }
      output->buf[i * N_ + j] = c;
    }
  }
}

/*
 * Softmax
 * Normalize the input elements according to its exp value.
 * The result can be interpreted as a probability distribution.
 * input: [*]
 * output: [*], (same shape as input)
 */
void softmax(Tensor *input, Tensor *output) {
  size_t n = input->num_elem();
  float sum = 0.0;
  for (size_t i = 0; i < n; i++) {
    float x = input->buf[i];
    sum += expf(x);
  }
  for (size_t i = 0; i < n; i++) {
    float x = input->buf[i];
    output->buf[i] = expf(x) / sum;
  }
}

/*
 * Sample a random index according to the given probability distribution
 * This function is called at most N*MAX_LEN times. Each call uses a
 * random float in [0,1] to sample an index from the given distribution.
 * input: [NUM_CHAR], probability distribution of the characters
 * rng_seq: [N*MAX_LEN],
 */
int random_select(Tensor *input, Tensor *rng_seq, int rng_offset) {
  float r = rng_seq->buf[rng_offset];
  size_t n = input->num_elem();
  float psum = 0.0;
  for (size_t i = 0; i < n; i++) {
    psum += input->buf[i];
    if (psum > r) {
      return i;
    }
  }
  return n - 1;
}

/*
 * Initialize the model.
 * Do input-independent job here.
 */
void namegen_initialize(int N, char *parameter_fname) {

  /* Only the root process reads the parameter */
 
  size_t parameter_binary_size = 0;
  float *parameter =
      (float *)read_binary(parameter_fname, &parameter_binary_size);

  /* Network parameters */
  character_embedding =
      new Tensor({NUM_CHAR, EMBEDDING_DIM}, parameter + OFFSET0);

  W_ir0 = new Tensor({HIDDEN_DIM, EMBEDDING_DIM}, parameter + OFFSET1);
  W_iz0 = new Tensor({HIDDEN_DIM, EMBEDDING_DIM}, parameter + OFFSET2);
  W_in0 = new Tensor({HIDDEN_DIM, EMBEDDING_DIM}, parameter + OFFSET3);
  W_ir1 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET4);
  W_iz1 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET5);
  W_in1 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET6);

  W_hr0 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET7);
  W_hz0 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET8);
  W_hn0 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET9);
  W_hr1 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET10);
  W_hz1 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET11);
  W_hn1 = new Tensor({HIDDEN_DIM, HIDDEN_DIM}, parameter + OFFSET12);

  b_ir0 = new Tensor({HIDDEN_DIM}, parameter + OFFSET13);
  b_iz0 = new Tensor({HIDDEN_DIM}, parameter + OFFSET14);
  b_in0 = new Tensor({HIDDEN_DIM}, parameter + OFFSET15);
  b_ir1 = new Tensor({HIDDEN_DIM}, parameter + OFFSET16);
  b_iz1 = new Tensor({HIDDEN_DIM}, parameter + OFFSET17);
  b_in1 = new Tensor({HIDDEN_DIM}, parameter + OFFSET18);

  b_hr0 = new Tensor({HIDDEN_DIM}, parameter + OFFSET19);
  b_hz0 = new Tensor({HIDDEN_DIM}, parameter + OFFSET20);
  b_hn0 = new Tensor({HIDDEN_DIM}, parameter + OFFSET21);
  b_hr1 = new Tensor({HIDDEN_DIM}, parameter + OFFSET22);
  b_hz1 = new Tensor({HIDDEN_DIM}, parameter + OFFSET23);
  b_hn1 = new Tensor({HIDDEN_DIM}, parameter + OFFSET24);

  W_fc = new Tensor({NUM_CHAR, HIDDEN_DIM}, parameter + OFFSET25);
  b_fc = new Tensor({NUM_CHAR}, parameter + OFFSET26);

  /* input, activations, output, etc. */
  input = new Tensor({1, N});
  emb_out = new Tensor({EMBEDDING_DIM, N});

  hidden0 = new Tensor({HIDDEN_DIM, N}, true);
  hidden1 = new Tensor({HIDDEN_DIM, N}, true);

  r0 = new Tensor({HIDDEN_DIM, N});
  z0 = new Tensor({HIDDEN_DIM, N});
  n0 = new Tensor({HIDDEN_DIM, N});

  r1 = new Tensor({HIDDEN_DIM, N});
  z1 = new Tensor({HIDDEN_DIM, N});
  n1 = new Tensor({HIDDEN_DIM, N});

  f = new Tensor({HIDDEN_DIM, N});

  rfloats = new Tensor({N, MAX_LEN});
  // ftmp0 = new Tensor({NUM_CHAR, N});
  char_prob = new Tensor({NUM_CHAR, N});
  hipDeviceSynchronize();
  printf("************ Initialization Success!!\n");
}

__global__ void fill_gpu_value(const int N, float *buf_gpu, const float _value){
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tidx >= N) return;
    buf_gpu[tidx] = _value;
    // printf("************ %f %d \n", buf_gpu[tidx], tidx);
  }

__global__ void gpu_embedding(const int N, const float *i_buf_gpu, 
                              const float *w_buf_gpu, float *o_buf_gpu){
  int _r = blockIdx.y * blockDim.y + threadIdx.y;
  int _c = blockIdx.x * blockDim.x + threadIdx.x;

  // if (r>= EMBEDDING_DIM || c >= N) return;
  // TODO: vernerable due to the typecasting
  // o_buf_gpu[r * N + c] = w_buf_gpu[(int)(i_buf_gpu[r]) * N + c];
  o_buf_gpu[_r*N + _c] = w_buf_gpu[(int)(i_buf_gpu[_c]) * EMBEDDING_DIM + _r];
}

__global__ void gpu_mmbmmbs(const int N, const int K1, const int K2,
                           float* _W1, float* _X1, float* _b1,
                           float* _W2, float* _X2, float* _b2,
                           float* _output
                           ){
int _r = blockIdx.y * blockDim.y + threadIdx.y;
int _c = blockIdx.x * blockDim.x + threadIdx.x;

float _sum = 0.0;
// if () return;

//TODO: Check!! is ++K right?
// // #pragma unroll 128
for (int k1=0; k1<K1; ++k1) 
{_sum += _W1[_r * K1 + k1] * _X1[k1 * N + _c];}

// // #pragma unroll 128
for (int k2=0; k2<K2; ++k2) 
{_sum += _W2[_r * K2 + k2] * _X2[k2 * N + _c];}
_sum += _b1[_r] + _b2[_r];
_output[_r * N + _c] += 1.0 / (1.0 + expf(-_sum));
}

__global__ void gpu_mmbrmmbt(const int N, const int K1, const int K2,
                           float* _W1, float* _X1, float* _b1,
                           float* _W2, float* _X2, float* _b2,
                           float* _r1, float* _output
                           ){
int _r = blockIdx.y * blockDim.y + threadIdx.y;
int _c = blockIdx.x * blockDim.x + threadIdx.x;

float _sum = 0.0;

//TODO: Check!! is ++K right?
// #pragma unroll 128
for (int _k=0; _k<K2; ++_k) 
{_sum += _W2[_r * K2 + _k] * _X2[_k * N + _c];}
_sum *= _r1[_c];
// #pragma unroll 128
for (int _k=0; _k<K1; ++_k) 
{_sum += _W1[_r * K1 + _k] * _X1[_k * N + _c];}

_sum += _b1[_r] + _b2[_r];
_output[_r * N + _c] += _gpu_tanh(_sum);
}

__global__ void gpu_compute_h(const int N, float* zt, float* nt,
                              float* ht
                           ){
int r = blockIdx.y * blockDim.y + threadIdx.y;
int c = blockIdx.x * blockDim.x + threadIdx.x;

// float sum = 0.0;
// TODO: this function is too simple we can reduce the number of threads
float _zt = zt[r*N + c];
ht[r*N + c] = (1-_zt) * nt[r*N + c] + _zt * ht[r*N + c];
}

__global__ void gpu_linear(const int N, const int K,
                           float* _W, float* _X, float* _b,
                           float* _output
                           ){
int _r = blockIdx.y * blockDim.y + threadIdx.y;
int _c = blockIdx.x * blockDim.x + threadIdx.x;

float _sum = 0.0;

//TODO: Check!! is ++K right?
// #pragma unroll 128
for (int _k=0; _k<K; ++_k) 
{_sum += _W[_r * K + _k] * _X[_k * N + _c];}
_sum += _b[_r];
_output[_r * N + _c] += _sum;
}

__global__ void softmax_kernel(float *input, float *output, int N) {
  // Calculate element-wise exponential and store in shared memory
  __shared__ float L[NUM_CHAR];

  unsigned int tid = threadIdx.x; // local
  unsigned int bn = blockIdx.x; // block number
  L[tid] = 0.0;
  L[tid] = expf(input[tid * N + bn]);

  __syncthreads();
  float _sum = 0.0;
  if (tid==0){
    for (int i=0; i<NUM_CHAR; i++){
      _sum += L[i];
    }
   }

  __syncthreads();
  output[tid * N + bn] = input[tid * N + bn] / _sum;  
}

__global__ void gpu_random_select(int N, int ll, float* cp, float* rf, 
char* output, float* input){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >=N) return;
  float psum = 0.0;
  for (int i=0; i< NUM_CHAR; i++){
    psum += cp[i*N + tid];
    if (psum > rf[tid * MAX_LEN + ll]){
      output[ll * (MAX_LEN+1) + tid] = (char)i;
      input[tid] = (float)i;
      return ;
    }
  }
  output[ll * (MAX_LEN+1) + tid] = (char)(NUM_CHAR-1);
  input[tid] = (float)(NUM_CHAR - 1);
}

float* _mat_to_print;

/*
 * Generate names.
 * Any input-dependent computation/communication must be done here.
 * N: # of names to generate
 * random_floats: N*MAX_LEN sequence of random floats in [0,1].
 * output: 2D-array of size N x (MAX_LEN+1), allocaetd at main.cpp
 */
void namegen(int N, float *random_floats, char *output) {

  // memcpy(rfloats->buf, random_floats, N * MAX_LEN * sizeof(float));
  CHECK_CUDA(hipMemcpy(rfloats->buf_gpu, random_floats, N * MAX_LEN * sizeof(float),
                        hipMemcpyHostToDevice));
  hipDeviceSynchronize();
  CHECK_CUDA(hipGetLastError());
  char* g_output;
  memset(output, 0, N * (MAX_LEN + 1) * sizeof(char));
  CHECK_CUDA(hipMalloc(&g_output, N * (MAX_LEN + 1) * sizeof(char)));
  hipDeviceSynchronize();
  CHECK_CUDA(hipGetLastError());

  /* Generate N names */
  /* Initialize input and hidden vector. */
  /* One hidden vector for each GRU layer */
  // input->buf[0] = SOS;
  // hidden0->set_zero();
  // hidden1->set_zero();
  dim3 gridDim_1((N+1023) / 1024);
  dim3 blockDim_1(1024);
  fill_gpu_value<<<gridDim_1, blockDim_1>>>(N, input->buf_gpu, (float)SOS);
  hipDeviceSynchronize();
  CHECK_CUDA(hipGetLastError());

  for (int l = 0; l < MAX_LEN; l++) {
    /* Embedding */
    // embedding(input, character_embedding, emb_out);
    dim3 blockDim_2(32,32);
    dim3 gridDim_2( (N + 31)/32, (EMBEDDING_DIM+31)/32);
    gpu_embedding<<<gridDim_2, blockDim_2>>>(N, input->buf_gpu, character_embedding->buf_gpu,
                  emb_out->buf_gpu);    
    hipDeviceSynchronize();
    CHECK_CUDA(hipGetLastError());
    
    // //////////////////////////////////////////////
    // /* Layer 1: input : emb_out & hid: hidden 0*/
    // //////////////////////////////////////////////
    
    dim3 blockDim_3(32,32);
    dim3 gridDim_3( (N + 31)/32, (HIDDEN_DIM+31)/32);
    gpu_mmbmmbs<<<gridDim_3, blockDim_3>>>(N, EMBEDDING_DIM, HIDDEN_DIM,
                W_ir0->buf_gpu, emb_out->buf_gpu, b_ir0->buf_gpu,
                W_hr0->buf_gpu, hidden0->buf_gpu, b_hr0->buf_gpu,
                r0->buf_gpu);
    hipDeviceSynchronize();
    CHECK_CUDA(hipGetLastError());

    if (l==0){
      size_t _to_print_size = N * HIDDEN_DIM;
      CHECK_CUDA(hipHostMalloc(&_mat_to_print, sizeof(float) * _to_print_size));
      CHECK_CUDA(hipMemcpy(_mat_to_print, r0->buf_gpu, 
      sizeof(float) * _to_print_size, hipMemcpyDeviceToHost));
      printf("\n");
      for (int _j =10; _j<15; _j++){
        for (int _i=10; _i < 15; _i++){
          printf("%f ,", _mat_to_print[_i *N + _j]);
        }
        printf("\n");
      }
    }

    // dim3 blockDim_4(32,32);
    // dim3 gridDim_4( (N + 31)/32, (HIDDEN_DIM+31)/32);
    
    gpu_mmbmmbs<<<gridDim_3, blockDim_3>>>(N, EMBEDDING_DIM, HIDDEN_DIM,
                W_iz0->buf_gpu, emb_out->buf_gpu, b_iz0->buf_gpu,
                W_hz0->buf_gpu, hidden0->buf_gpu, b_hz0->buf_gpu,
                z0->buf_gpu);
    hipDeviceSynchronize();
    CHECK_CUDA(hipGetLastError());

    gpu_mmbrmmbt<<<gridDim_3, blockDim_3>>>(N, EMBEDDING_DIM, HIDDEN_DIM,
                W_in0->buf_gpu, emb_out->buf_gpu, b_in0->buf_gpu,
                W_hn0->buf_gpu, hidden0->buf_gpu, b_hn0->buf_gpu,
                r0->buf_gpu, n0->buf_gpu);
    hipDeviceSynchronize();
    CHECK_CUDA(hipGetLastError());

//     //TODO: is it able to overwrite hidden0?
  gpu_compute_h<<<gridDim_3, blockDim_3>>>(N, z0->buf_gpu, n0->buf_gpu,
                                            hidden0->buf_gpu);
  hipDeviceSynchronize();
  CHECK_CUDA(hipGetLastError());
//     //////////////////////////////////////////////
//     /* Layer 2: input : hidden0 & hid: hidden 1*/
//     //////////////////////////////////////////////

    gpu_mmbmmbs<<<gridDim_3, blockDim_3>>>(N, HIDDEN_DIM, HIDDEN_DIM,
                W_ir1->buf_gpu, hidden0->buf_gpu, b_ir1->buf_gpu,
                W_hr1->buf_gpu, hidden1->buf_gpu, b_hr1->buf_gpu,
                r1->buf_gpu);

    gpu_mmbmmbs<<<gridDim_3, blockDim_3>>>(N, HIDDEN_DIM, HIDDEN_DIM,
                W_iz1->buf_gpu, hidden0->buf_gpu, b_iz1->buf_gpu,
                W_hz1->buf_gpu, hidden1->buf_gpu, b_hz1->buf_gpu,
                z1->buf_gpu);

    gpu_mmbrmmbt<<<gridDim_3, blockDim_3>>>(N, HIDDEN_DIM, HIDDEN_DIM,
                W_in1->buf_gpu, hidden0->buf_gpu, b_in1->buf_gpu,
                W_hn1->buf_gpu, hidden1->buf_gpu, b_hn1->buf_gpu,
                r1->buf_gpu, n1->buf_gpu);

    //TODO: is it able to overwrite hidden0?
    gpu_compute_h<<<gridDim_3, blockDim_3>>>(N, z1->buf_gpu, n1->buf_gpu,
                                              hidden1->buf_gpu);
  hipDeviceSynchronize();
  CHECK_CUDA(hipGetLastError());
    
//     //////////////////////////////////////////////
//     /* Linear: input : hidden1                  */
//     //////////////////////////////////////////////
gpu_linear<<<gridDim_3, blockDim_3>>>(N, HIDDEN_DIM,
W_fc->buf_gpu, hidden1->buf_gpu, b_fc->buf_gpu,f->buf_gpu);
softmax_kernel<<<N, NUM_CHAR>>>(f->buf_gpu, char_prob->buf_gpu, N);

//     //////////////////////////////////////////////
//     /* Move results to CPU and finalize         */
//     //////////////////////////////////////////////

    dim3 blockDim_rand(1024);
    dim3 gridDim_rand((N+1023)/1024);
    gpu_random_select<<<gridDim_rand, blockDim_rand>>>(
      N, l, char_prob->buf_gpu, rfloats->buf_gpu, g_output, input->buf_gpu
    );
  hipDeviceSynchronize();
  CHECK_CUDA(hipGetLastError());
  }
hipDeviceSynchronize();
CHECK_CUDA(hipGetLastError());
CHECK_CUDA(hipMemcpy(output, g_output, N * (MAX_LEN+1) * sizeof(char),
                        hipMemcpyDeviceToHost));
hipDeviceSynchronize();
CHECK_CUDA(hipGetLastError());

}

/*
 * Finalize the model.
 * Although it is not neccessary, we recommend to deallocate and destruct
 * everything you made in namegen_initalize() and namegen().
 */
void namegen_finalize() {

  delete character_embedding;
  delete W_ir0;
  delete W_iz0;
  delete W_in0;
  delete W_ir1;
  delete W_iz1;
  delete W_in1;
  delete W_hr0;
  delete W_hz0;
  delete W_hn0;
  delete W_hr1;
  delete W_hz1;
  delete W_hn1;
  delete b_ir0;
  delete b_iz0;
  delete b_in0;
  delete b_ir1;
  delete b_iz1;
  delete b_in1;
  delete b_hr0;
  delete b_hz0;
  delete b_hn0;
  delete b_hr1;
  delete b_hz1;
  delete b_hn1;
  delete W_fc;
  delete b_fc;
  delete rfloats;

  delete input;
  delete emb_out;
  delete hidden0;
  delete hidden1;
  delete r0;
  delete r1;
  delete z0;
  delete z1;
  delete n0;
  delete n1;
  delete f;
  delete char_prob;
  delete rtmp00;
  delete rtmp01;
  delete rtmp02;
  delete rtmp03;
  delete rtmp04;
  delete rtmp10;
  delete rtmp11;
  delete rtmp12;
  delete rtmp13;
  delete rtmp14;
  delete ztmp00;
  delete ztmp01;
  delete ztmp02;
  delete ztmp03;
  delete ztmp04;
  delete ztmp10;
  delete ztmp11;
  delete ztmp12;
  delete ztmp13;
  delete ztmp14;
  delete ntmp00;
  delete ntmp01;
  delete ntmp02;
  delete ntmp03;
  delete ntmp04;
  delete ntmp05;
  delete ntmp10;
  delete ntmp11;
  delete ntmp12;
  delete ntmp13;
  delete ntmp14;
  delete ntmp15;
  delete htmp00;
  delete htmp01;
  delete htmp02;
  delete htmp10;
  delete htmp11;
  delete htmp12;
  delete ftmp0;
}